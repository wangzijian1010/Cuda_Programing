
#include <hip/hip_runtime.h>
#include <iostream>

// 循环的并行化
__global__ void some_kernal_func(int* const a,int* const b,int* const c){
    const unsigned int thread_idx = threadIdx.x;
    a[thread_idx] = b[thread_idx] * c[thread_idx];
}

int main() {
    std::cout << "Hello, World!" << std::endl;
    return 0;
}
